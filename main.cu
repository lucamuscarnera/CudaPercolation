#include <iostream>
#include "headers/Lattice.h"

int main()
{
	Settings base_settings;
	base_settings.summary();
	Lattice lattice(42, 2048, base_settings);
	// run the simulation
	lattice.observe(0.59);

	std::cout << "End" << std::endl;
}
